#include "hip/hip_runtime.h"
/* Author Alfredo Daniel Sanchez: alfredo.daniel.sanchez@gmail.com */

#include <iostream>
#include <string>
#include <random>
#include <fstream>
#include <iomanip>
#include <typeinfo>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <stdio.h>
#include <chrono>
#include <sys/time.h>
#include <hipfft/hipfft.h>

#include <fftw3.h>

// Complex data type
using complex_t = hipfftComplex;
using real_t = float;

// Define constants
const real_t PI   = 3.14159265358979323846;    // pi
const real_t C    = 299792458*1E6/1E12;        // speed of ligth in vacuum [um/ps]
const real_t EPS0 = 8.8541878128E-12*1E12/1E6; // vacuum pertivity [W.ps/V²μm] 

const uint SIZE   = 1 << 12; // vector size
const uint NZ     = 150;     // steps over z direction
const uint NRT    = 500;   // number of round trips 

// Package headers
#include "headers/common.h"
#include "headers/operators.h"
#ifdef PPLN // Mgo:PPLN nonlinear crystal
#include "headers/ppln.h"
#endif
#ifdef SPPLT // Mgo:sPPLT nonlinear crystal
#include "headers/spplt.h"
#endif
#include "headers/functions.h"
#ifdef THREE_EQS // Define 2 or 3 coupled-wave equations
#include "headers/cwes3.h"
#else
#include "headers/cwes2.h"
#endif
#include "headers/files.h"



int main(int argc, char *argv[]){
	
	////////////////////////////////////////////////////////////////////////////////////////
	// Set GPU and timing
	
	std::cout << "\n\n\n#######---Welcome to OPO calculator---#######\n\n\n" << std::endl;
	
	time_t current_time; // timing the code
	time(&current_time);
	std::cout << "Date: " << ctime(&current_time) << std::endl;
	double iStart = seconds();
	
	
	#ifdef CW_OPO
	std::cout << "Regime: continuous wave" << std::endl;
	#endif
	#ifdef NS_OPO
	std::cout << "Regime: nanosecond" << std::endl;
	#endif
	#ifdef THREE_EQS
	std::cout << "Three equations" << std::endl;
	#else
	std::cout << "Two equations" << std::endl;
	#endif
	
	////////////////////////////////////////////////////////////////////////////////////////
	
	
	
	
	////////////////////////////////////////////////////////////////////////////////////////
	//* Define simulation parameters, physical quantities and set electric fields */
	
	// Grids, crystal and cavity parameters //
	real_t lp        = atof(argv[1])*1e-3;  // pump wavelength   [μm]
	real_t ls        = 1.060; //2*lp;           // signal wavelength [μm]
	real_t li        = lp*ls/(ls-lp);  // idler wavelength  [μm]
	
	// 	real_t deff      = 14.77e-6;       // effective second-order susceptibility [um/V]
	real_t Temp      = atof(argv[2]); // crystal temperature [ºC]
	real_t Lambda    = atof(argv[3]); // grating period for QPM [μm]  
	real_t Lcr       = 5e3;            // crystal length [um]
	
	real_t np        = n(lp, Temp);         // pump ref. index
	real_t vp        = group_vel(lp, Temp); // pump group velocity [μm/ps]
	real_t b2p       = gvd(lp, Temp);       // pump GVD [ps²/μm] 
	real_t b3p       = 0.*TOD(lp, Temp);    // pump TOD [ps³/μm]	
	real_t kp        = 2*PI*deff/(np*lp);        // pump kappa [1/V]
	
	real_t ns        = n(ls, Temp);         // signal ref. index
	real_t vs        = group_vel(ls, Temp); // signal group velocity [μm/ps]
	real_t b2s       = gvd(ls, Temp);       // signal GVD [ps²/μm] 
	real_t b3s       = 0.*TOD(ls, Temp);    // signal TOD [ps³/μm]		
	real_t ks        = 2*PI*deff/(ns*ls);        // signal kappa [1/V]
	
	real_t ni        = n(li, Temp);         // idler ref. index
	real_t vi        = group_vel(li, Temp); // idler group velocity [μm/ps]
	real_t b2i       = gvd(li, Temp);       // idler GVD [ps²/μm]
	real_t b3i       = 0.*TOD(li, Temp);    // idler TOD [ps³/μm]	
	real_t ki        = 2*PI*deff/(ni*li);        // idler kappa [1/V]
	
	real_t dk        = 2*PI*( np/lp-ns/ls-ni/li-1/Lambda ); // mismatch factor
	real_t dkp       = 1/vp-1/vs;                           // group velocity mismatch	
	real_t Lcav      = 5 * Lcr;                             // cavity length [um]
	real_t Rs        = atof(argv[5])*0.01;                  // Reflectivity at signal wavelength 
	real_t alphas    = 0.5*((1-Rs)+alpha_crs*Lcr);          // Total losses for threshold condition signal
	#ifdef THREE_EQS
	real_t Ri        = 0.98;                                 // Reflectivity at idler wavelength 
	real_t alphai    = 0.5*((1-Ri)+alpha_cri*Lcr);          // Total losses for threshold condition idler
	#endif
	real_t t_rt      = (Lcav+Lcr*(ns-1))/C;                 // round-trip time [ps]
	real_t FSR       = 1/t_rt;	                          // free-spectral range
	real_t finesse   = 2*PI/(1-Rs);                         // cavity finesse
	real_t lw        = FSR/finesse*1e6;                     // cavity Linewidth [MHz]
	real_t delta     = atof(argv[6]);                       // cavity detuning [rad] 
	real_t epsilon   = atof(argv[7])*0.01;                  // dispersion compensation index
	real_t GDD       = -epsilon*b2s*Lcr;                    // GDD [ps²]
	real_t TODscomp  = -0.01*atof(argv[8])*b3s*Lcr;        // TOD compensation [ps³]
	real_t TODicomp  = -0.01*atof(argv[8])*b3i*Lcr;        // TOD compensation [ps³]
	
	
	// z discretization, time and frequency discretization
	real_t dz        = Lcr/NZ;   // number of z-steps in the crystal
	real_t dT        = t_rt/SIZE; // time step in [ps]
	real_t dF        = 1/t_rt;    // frequency step in [THz]
	
	
	bool video       = false;
	uint Nrts;        // number of last round trips to save (only for cw)
	if(video){Nrts = 100;}
	else{Nrts = 16;}
	
	
	#ifdef CW_OPO
	uint SIZEL = SIZE*Nrts; // size of large vectors for full simulation
	real_t T_width = t_rt*Nrts; // total time for the saved simulation
	#endif
	#ifdef NS_OPO
	uint SIZEL = SIZE*NRT; // size of large vectors for full simulation
	real_t T_width = t_rt*NRT; // total time for the saved simulation
	#endif	
	
	// Time vector T for one round trip
	real_t *T = (real_t*) malloc(sizeof(real_t) * SIZE);
	linspace( T, SIZE, -0.5*t_rt, 0.5*t_rt);
	
	// Time vector Tp for full simulation
	real_t *Tp = (real_t*) malloc(sizeof(real_t) * SIZEL);
	linspace( Tp, SIZEL, -0.5*T_width, 0.5*T_width);
	
	// Time vector Fp for full simulation
	real_t dFp  = 1/T_width;
	real_t *Fp = (real_t*) malloc(sizeof(real_t) * SIZEL);
	linspace( Fp, SIZEL, -0.5*SIZEL*dFp, +0.5*SIZEL*dFp);
	
	// Frequency and angular frequency vectors f and Ω
	real_t *F = (real_t*) malloc(sizeof(real_t) * SIZE);
	linspace( F, SIZE, -0.5*SIZE*dF, +0.5*SIZE*dF);
	real_t *w = (real_t*) malloc(sizeof(real_t) * SIZE);
	fftshift(w,F, SIZE);
	for (uint i=0; i<SIZE; i++){
		w[i] = 2*PI*w[i]; // angular frequency [2*pi*THz]
	}
		
	// Define memory size for complex host vectors
	uint nBytes   = sizeof(complex_t)*SIZE;
	
	// Difine which fields are resonant (SRO, DRO or TRO)
	bool is_Ap_resonant = false;
	bool is_As_resonant = true;
	#ifdef THREE_EQS
	bool is_Ai_resonant = true;
	#endif
	
	// Define input pump parameters
	real_t waist = 55;             // beam waist radius [um]
	real_t spot  = PI*waist*waist; // spot area [μm²]
	real_t Ith, Pth;               // Power and intensity threshold 
	#ifdef THREE_EQS
	// Power and intensity threshold non-degenerate OPO 
	if (!is_Ai_resonant){
		std::cout << "SRO: As is resonant" << std::endl;
		Ith   = EPS0*C*np*ns*ni*ls*li*pow((1/deff/Lcr/PI),2)*alphas/2;
	}
	if (!is_As_resonant){
		std::cout << "SRO, Ai is resonant" << std::endl;
		Ith   = EPS0*C*np*ns*ni*ls*li*pow((1/deff/Lcr/PI),2)*alphai/2;
	}
	if (is_As_resonant and is_Ai_resonant){
		std::cout << "DRO, As and Ai are resonant" << std::endl;
		Ith   = EPS0*C*np*ns*ni*ls*li*pow((1/deff/Lcr/PI),2)*alphas*alphai/8;
	}	
	Pth   = Ith*spot;
	#else
	// Power and intensity threshold degenerate DRO 
	Ith   = EPS0*C*np*powf((ns*ls*alphas/deff/Lcr/PI), 2)/8;
	Pth   = Ith*spot;
	#endif
	
	real_t Nth   = atof(argv[4]);             // Times over the threshold
	real_t Inten = atof(argv[4])*Ith;         // Pump intensity in [W/um²]
	real_t Power = Inten*spot;                // Pump power in [W]
	real_t Ap0   = sqrt(2*Inten/(np*EPS0*C)); // Input pump field strength [V/μm]
	
	// Define input pump vector
	#ifdef CW_OPO
	complex_t *Ap    = (complex_t*)malloc(nBytes); // input pump vector
	complex_t *Ap_in = (complex_t*)malloc(nBytes); // input pump vector
	input_field_T(Ap_in, Ap0, SIZE );              // set input pump vector (cw)
	#endif
	
	#ifdef NS_OPO
	real_t FWHM      = 10000;                              // intensity FWHM for input [ps]
	real_t sigmap    = FWHM*sqrtf(2)/(2*sqrtf(2*logf(2))); // σ of electric field gaussian pulse [ps]
	complex_t *Ap    = (complex_t*)malloc(nBytes); // input pump vector
	complex_t *Ap_in = (complex_t*)malloc(sizeof(complex_t)*SIZEL);// input pump vector
	input_field_T(Ap_in, Ap0, Tp, sigmap, SIZEL); // set input pump vector (gaussian pulse)
	#endif
	
	
	// Define input signal vector (NOISE)
	complex_t *As = (complex_t*)malloc(nBytes);
	NoiseGeneratorCPU ( As, SIZE );
	
	#ifdef THREE_EQS	
	// Define input idler vector (NOISE)
	complex_t *Ai = (complex_t*)malloc(nBytes);
	NoiseGeneratorCPU ( Ai, SIZE );
	#endif
	
	
	// Intracavy phase modulator
	bool using_phase_modulator = atoi(argv[9]);
	real_t mod_depth, fpm, df;
	if(using_phase_modulator){
		mod_depth       = atof(argv[10])*PI;
		df              = atof(argv[11])*sqrtf(Nth-1)*alphas/(PI*mod_depth)*FSR;
		fpm             = FSR - df;
	}
	
	
	// Define string variables for saving files
	std::string Filename, SAux, Extension = ".dat";
	bool save_input_fields = false;  // Save input fields files
	if (save_input_fields){
		#ifdef CW_OPO
		Filename = "pump_input";	SaveVectorComplex (Ap_in, SIZE, Filename);
		#endif
		#ifdef NS_OPO
		Filename = "pump_input";	SaveVectorComplex (Ap_in, SIZEL, Filename);
		#endif
		Filename = "signal_input";	SaveVectorComplex (As, SIZE, Filename);
		#ifdef THREE_EQS	
		Filename = "idler_input";	SaveVectorComplex (Ai, SIZE, Filename);	
		#endif
	}
	
	
	bool print_param_on_screen = true;	// Print parameters on screen
	if ( print_param_on_screen ){
		std::cout << "\n\nSimulation parameters:\n\n " << std::endl;
		std::cout << "Number of round trips   = " << NRT  << std::endl;
		std::cout << "Pump wavelength         = " << lp*1e3 << " nm" << std::endl;
		std::cout << "Signal wavelength       = " << ls*1e3 << " nm" << std::endl;
		#ifdef THREE_EQS
		std::cout << "Idler wavelength        = " << li*1e3 << " nm" << std::endl;
		#endif
		std::cout << "Temp                    = " << Temp << " ºC" << std::endl;
		std::cout << "np                      = " << np << std::endl;
		std::cout << "ns                      = " << ns << std::endl;
		std::cout << "ni                      = " << ni << std::endl;
		std::cout << "\u03BD⁻¹ pump                = " << 1.0/vp << " ps/\u03BCm" << std::endl;
		std::cout << "\u03BD⁻¹ signal              = " << 1.0/vs << " ps/\u03BCm" << std::endl;
		#ifdef THREE_EQS
		std::cout << "\u03BD⁻¹ idler               = " << 1.0/vi << " ps/\u03BCm" << std::endl;		
		#endif
		std::cout << "\u0394k                      = " << dk << " \u03BCm⁻¹" << std::endl;
		std::cout << "\u0394k'                     = " << dkp << " ps/\u03BCm" << std::endl;	
		std::cout << "GVD pump                = " << b2p << " ps²/\u03BCm" << std::endl;
		std::cout << "GVD signal              = " << b2s << " ps²/\u03BCm" << std::endl;
		#ifdef THREE_EQS
		std::cout << "GVD idler               = " << b2i << " ps²/\u03BCm" << std::endl;		
		#endif
		std::cout << "TOD pump                = " << b3p << " ps³/\u03BCm" << std::endl;
		std::cout << "TOD signal              = " << b3s << " ps³/\u03BCm" << std::endl;		
		std::cout << "Net GVD                 = " << (1-epsilon)*b2s << " ps²/\u03BCm" << std::endl;
		std::cout << "GVD compensation        = " << atoi(argv[7]) << " %"  << std::endl;
		std::cout << "Net TOD                 = " << (1-0.01*atoi(argv[8]))*b3s*Lcr*1e3 << " fs³"  << std::endl;
		std::cout << "TOD compensation        = " << atof(argv[8]) << " %"  << std::endl;		
		std::cout << "deff                    = " << deff*1e6 << " pm/V"  << std::endl;
		std::cout << "\u039B                       = " << Lambda << " \u03BCm"  << std::endl;
		std::cout << "\u03B1cp                     = " << alpha_crp << " \u03BCm⁻¹"  << std::endl;
		std::cout << "\u03B1cs                     = " << alpha_crs << " \u03BCm⁻¹" << std::endl;
		std::cout << "\u03B1s                      = " << alphas << std::endl;
		#ifdef THREE_EQS
		std::cout << "\u03B1ci                      = " << alpha_cri << " \u03BCm⁻¹" << std::endl;
		std::cout << "\u03B1i                      = " << alphas << std::endl;
		#endif
		
		std::cout << "Crystal length          = " << Lcr*1e-3 << " mm"  << std::endl;
		std::cout << "Cavity  length          = " << Lcav*1e-3 << " mm"  << std::endl;
		std::cout << "\u0394z                      = " << dz << " \u03BCm"  << std::endl;
		std::cout << "Reflectivity (signal)   = " << Rs*100 << " %"  << std::endl;
		#ifdef THREE_EQS
		std::cout << "Reflectivity (idler)    = " << Ri*100 << " %"  << std::endl;	
		#endif
		std::cout << "Cavity Finesse          = " << finesse << std::endl;	
		std::cout << "Cavity lw (FWHM)        = " << lw << " MHz"  << std::endl;	
		std::cout << "Round-trip time         = " << std::setprecision(15) << t_rt << " ps"  << std::endl;	
		std::cout << "FSR                     = " << std::setprecision(15) << FSR*1e3 << " GHz"  << std::endl;
		std::cout << "Cavity detuning (\u03B4)     = " << delta << "\u03C0"  << std::endl;	
		std::cout << "Using N                 = " << SIZE << " points" << std::endl;
		std::cout << "dT                      = " << dT << " ps" << std::endl;
		std::cout << "SIZEL                   = " << SIZEL << std::endl;
		std::cout << "dFp                     = " << dFp << " THz" << std::endl;
		std::cout << "Max frequency           = " << Fp[SIZEL-1] << " THz" << std::endl;
		std::cout << "Ap0                     = " << Ap0 << " V/um" << std::endl; 
		std::cout << "waist                   = " << waist << " \u03BCm" << std::endl;
		std::cout << "spot                    = " << spot << " \u03BCm²" << std::endl;
		std::cout << "Power threshold         = " << Pth << " W" << std::endl;
		std::cout << "Power                   = " << Power << " W" << std::endl;
		std::cout << "Times above the thres.  = " << Nth << std::endl;
		if(using_phase_modulator){
			std::cout << "Using a phase modulator:" << std::endl;
			std::cout << "Mod. depth (\u03B2)          = " << atof(argv[10]) << "\u03C0 rad = " << mod_depth << " rad" << std::endl;
			std::cout << "Freq. detuning (\u03B4f)     = " << df*1e6 << " MHz" << std::endl;
			std::cout << "Mod. frequency(fm)      = " << fpm*1e3 << " GHz" << std::endl;
			std::cout << "\n\nPoint in the space of parameters:\n" << std::endl;
			std::cout << "(N,\u03B2,\u03B4f,\u03B5) = ( " << Nth << ", " << atof(argv[10]) << ", "  << std::setprecision(4) << df*1e6 << ", " << epsilon << " )\n\n" << std::endl;			
		}
		else{std::cout << "No phase modulator" << std::endl;
			std::cout << "\n\nPoint in the space of parameters:\n" << std::endl;
			std::cout << "( N, \u03B2, \u03B4f, \u03B5 ) = ( " << Nth << ", 0, 0, " << std::setprecision(2) << epsilon << " )\n\n" << std::endl;
		}
	}
	////////////////////////////////////////////////////////////////////////////////////////
	
	
	
	
	////////////////////////////////////////////////////////////////////////////////////////
	//* Define CPU vectors //

	complex_t *As_total = (complex_t*)malloc(sizeof(complex_t)*SIZEL);
	complex_t *Ap_total = (complex_t*)malloc(sizeof(complex_t)*SIZEL);
	complex_t *Asw = (complex_t*)malloc(nBytes);
	complex_t *Apw = (complex_t*)malloc(nBytes);

	
	// RK4 (kx) and auxiliary (aux) CPU vectors 
	complex_t *k1p = (complex_t*)malloc(nBytes);
	complex_t *k2p = (complex_t*)malloc(nBytes);
	complex_t *k3p = (complex_t*)malloc(nBytes);
	complex_t *k4p = (complex_t*)malloc(nBytes);
	complex_t *k1s = (complex_t*)malloc(nBytes);
	complex_t *k2s = (complex_t*)malloc(nBytes);
	complex_t *k3s = (complex_t*)malloc(nBytes);
	complex_t *k4s = (complex_t*)malloc(nBytes);
	complex_t *auxp = (complex_t*)malloc(nBytes);
	complex_t *auxs = (complex_t*)malloc(nBytes);
	

	#ifdef THREE_EQS
	complex_t *Aiw = (complex_t*)malloc(nBytes);
	complex_t *Ai_total = (complex_t*)malloc(sizeof(complex_t)*SIZEL);
	complex_t *k1i = (complex_t*)malloc(nBytes);
	complex_t *k2i = (complex_t*)malloc(nBytes);
	complex_t *k3i = (complex_t*)malloc(nBytes);
	complex_t *k4i = (complex_t*)malloc(nBytes);
	complex_t *auxi = (complex_t*)malloc(nBytes);
	bool idler_pm = true; // phase modulator applies on idler
	#endif
	
	////////////////////////////////////////////////////////////////////////////////////////
	
	
	
	
	////////////////////////////////////////////////////////////////////////////////////////	
	//* Main loop. Fields in the cavity */
	
	// Set plan for FFT //
	fftwf_plan plan = NULL; // c2c for input field

	std::cout << "Starting main loop on CPU & GPU...\n" << std::endl;
	uint mm = 0; // counts for cw saved round trips
	for (uint nn = 0; nn < NRT; nn++){
		if( nn%10 == 0 or nn == NRT-1 )
			std::cout << "#round trip: " << nn << std::endl;
		
		#ifdef CW_OPO
		// update the input pump in each round trip
		memcpy( Ap, Ap_in, nBytes );
		#endif
		#ifdef NS_OPO
		// read the input pump in nn-th round trip
		ReadPump ( Ap, Ap_in, NRT, nn, SIZE );
		#endif
		
		#ifdef THREE_EQS
		if (!is_Ai_resonant){	// For non-resonant field, it resets Ai in every round trip
			NoiseGeneratorCPU ( Ai, SIZE );
		}
		#endif
		
		if (!is_As_resonant){	// For non-resonant field, it resets As in every round trip
			NoiseGeneratorCPU ( As, SIZE );
		}
		
		#ifdef THREE_EQS // Single pass for coupled wave equations (2 or 3)
		EvolutionInCrystal( w, Ap, As, Ai, Apw, Asw, Aiw, k1p, k1s, k1i, k2p, k2s, k2i, k3p, k3s, k3i, k4p, k4s, k4i, auxp, auxs, auxi, lp, ls, li, vp, vs, vi, b2p, b2s, b2i, b3p, b3s, b3i, dk, kp, ks, ki, dz );
		#else
		EvolutionInCrystal( w, Ap, As, Apw, Asw, k1p, k1s, k2p, k2s, k3p, k3s, k4p, k4s, auxp, auxs, lp, ls, vp, vs, b2p, b2s, b3p, b3s, dk, kp, ks, dz );
		#endif
		
		
		if(GDD!=0){ // adds dispersion compensation
			AddGDD (Asw, auxs, w, GDD);
			plan = fftwf_plan_dft_1d(SIZE, reinterpret_cast<fftwf_complex*>(Asw), reinterpret_cast<fftwf_complex*>(As), FFTW_FORWARD, FFTW_MEASURE);
			fftwf_execute(plan);
			#ifdef THREE_EQS
			AddGDD (Aiw, auxi, w, GDD);
			plan = fftwf_plan_dft_1d(SIZE, reinterpret_cast<fftwf_complex*>(Aiw), reinterpret_cast<fftwf_complex*>(Ai), FFTW_FORWARD, FFTW_MEASURE);
			fftwf_execute(plan);

			#endif
		}		
		
		if( using_phase_modulator ){ // use an intracavy phase modulator of one o more fields
			PhaseModulatorIntraCavity(As, auxs, mod_depth, fpm, T);
			#ifdef THREE_EQS
			if(idler_pm){
				PhaseModulatorIntraCavity(Ai, auxi, mod_depth, fpm, T);
			}
			#endif
		}
		
		if (is_As_resonant){ // if As is resonant, adds phase and losses
			AddPhase(As, auxs, Rs, delta, nn);
		}
		
		#ifdef THREE_EQS
		if (is_Ai_resonant){  // if Ai is resonant, adds phase and losses
			AddPhase(Ai, auxi, Ri, delta, nn);
		}
		#endif
		
		#ifdef CW_OPO	// saves systematically every round trip
		if (video){  
			if (nn % 100 == 0){ // this branch is useful if the user want to save the round trips every 100 ones
				std::cout << "Saving the " << nn << "-th round trip" << std::endl;
				SaveRoundTrip(As_total, As, mm, Nrts ); // saves signal
				SaveRoundTrip(Ap_total, Ap, mm, Nrts ); // saves pump
				#ifdef THREE_EQS
				SaveRoundTrip(Ai_total, Ai, mm, Nrts ); // saves idler
				#endif
				mm += 1;
			}			
		}
		else{  // this branch is useful if the user want to save the last NRT-Nrts round trips
			if (nn >= NRT -Nrts){                
				SaveRoundTrip( Ap_total, Ap, mm, Nrts ); // saves pump
				SaveRoundTrip( As_total, As, mm, Nrts ); // saves signal
				#ifdef THREE_EQS
				SaveRoundTrip( As_total, As, mm, Nrts ); // saves idler
				#endif
				mm += 1;
			}
		}
		#endif
		#ifdef NS_OPO	// save the simulation in the NS regime
		SaveRoundTrip(Ap_total, Ap, nn, NRT ); // saves pump
		SaveRoundTrip(As_total, As, mm, NRT ); // saves signal
		#ifdef THREE_EQS
		SaveRoundTrip(Ai_total, Ai, mm, NRT ); // saves idler
		#endif
		#endif
		
	} // End of main loop
	
	////////////////////////////////////////////////////////////////////////////////////////
	
	
	
	
	////////////////////////////////////////////////////////////////////////////////////////	
	// Saving results in .dat files using the function SaveFileVectorComplex() //
	 
	bool save_vectors = true; // Decide whether or not save these vectors
	if (save_vectors){
		std::cout << "\nSaving time and frequency vectors...\n" << std::endl;
		Filename = "Tp"; SaveVectorReal (Tp, SIZEL, Filename+Extension);
		Filename = "freq"; SaveVectorReal (Fp, SIZEL, Filename+Extension);
		Filename = "T"; SaveVectorReal (T, SIZE, Filename+Extension);
	}
	else{ std::cout << "\nTime and frequency were previuosly save...\n" << std::endl;
	}
	
	// Save the full simulation
	Filename = "signal_output";	SaveVectorComplex ( As_total, SIZEL, Filename );
	Filename = "pump_output";	SaveVectorComplex ( Ap_total, SIZEL, Filename );
	#ifdef THREE_EQS
	Filename = "idler_output";	SaveVectorComplex ( Ai_total, SIZEL, Filename );
	#endif
	
	////////////////////////////////////////////////////////////////////////////////////////
	
	
	
	
	////////////////////////////////////////////////////////////////////////////////////////
	// Deallocating memory from CPU and destroying plans
	
	free(Tp); free(T); free(Fp); free(w); free(F);
	free(Ap); free(As); free(Ap_in);
	free(k1p); free(k2p); free(k3p); free(k4p);
	free(k1s); free(k2s); free(k3s); free(k4s);
	free(auxp); free(auxs);
	
	#ifdef THREE_EQS
	free(Ai); free(Ai_total);
	free(k1i); free(k2i); free(k3i); free(k4i);
	free(auxi);
	#endif
	
	// Destroy FFT context
	fftwf_destroy_plan(plan); 
	
	////////////////////////////////////////////////////////////////////////////////////////
	
	
	
	
	////////////////////////////////////////////////////////////////////////////////////////	
	// Finish timing: returns the runtime simulation
	
	double iElaps = seconds() - iStart;
	if(iElaps>60){std::cout << "\n\n...time elapsed " <<  iElaps/60.0 << " min\n\n " << std::endl;}
	else{std::cout << "\n\n...time elapsed " <<  iElaps << " seconds\n\n " << std::endl;}
	
	time(&current_time);
	std::cout << ctime(&current_time) << std::endl;
	
	return 0;
}
